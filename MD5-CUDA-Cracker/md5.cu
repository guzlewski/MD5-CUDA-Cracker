﻿#include "md5.cuh"

__device__ constexpr uchar padding[block_size] = { 0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

__device__ __host__ uint byteswap(uint word)
{
	return ((word >> 24) & 0x000000FF) | ((word >> 8) & 0x0000FF00) | ((word << 8) & 0x00FF0000) | ((word << 24) & 0xFF000000);
}

__device__ __host__ void transform(uint state[4], const uchar block[block_size])
{
	uint a = state[0], b = state[1], c = state[2], d = state[3];
	uint x[16];

	for (uint i = 0, j = 0; j < block_size && i < 16; i++, j += 4)
	{
		x[i] = (uint)block[j] | ((uint)block[j + 1] << 8) |
			((uint)block[j + 2] << 16) | ((uint)block[j + 3] << 24);
	}

	FF(a, b, c, d, x[0], S11, 0xd76aa478);
	FF(d, a, b, c, x[1], S12, 0xe8c7b756);
	FF(c, d, a, b, x[2], S13, 0x242070db);
	FF(b, c, d, a, x[3], S14, 0xc1bdceee);
	FF(a, b, c, d, x[4], S11, 0xf57c0faf);
	FF(d, a, b, c, x[5], S12, 0x4787c62a);
	FF(c, d, a, b, x[6], S13, 0xa8304613);
	FF(b, c, d, a, x[7], S14, 0xfd469501);
	FF(a, b, c, d, x[8], S11, 0x698098d8);
	FF(d, a, b, c, x[9], S12, 0x8b44f7af);
	FF(c, d, a, b, x[10], S13, 0xffff5bb1);
	FF(b, c, d, a, x[11], S14, 0x895cd7be);
	FF(a, b, c, d, x[12], S11, 0x6b901122);
	FF(d, a, b, c, x[13], S12, 0xfd987193);
	FF(c, d, a, b, x[14], S13, 0xa679438e);
	FF(b, c, d, a, x[15], S14, 0x49b40821);

	GG(a, b, c, d, x[1], S21, 0xf61e2562);
	GG(d, a, b, c, x[6], S22, 0xc040b340);
	GG(c, d, a, b, x[11], S23, 0x265e5a51);
	GG(b, c, d, a, x[0], S24, 0xe9b6c7aa);
	GG(a, b, c, d, x[5], S21, 0xd62f105d);
	GG(d, a, b, c, x[10], S22, 0x2441453);
	GG(c, d, a, b, x[15], S23, 0xd8a1e681);
	GG(b, c, d, a, x[4], S24, 0xe7d3fbc8);
	GG(a, b, c, d, x[9], S21, 0x21e1cde6);
	GG(d, a, b, c, x[14], S22, 0xc33707d6);
	GG(c, d, a, b, x[3], S23, 0xf4d50d87);
	GG(b, c, d, a, x[8], S24, 0x455a14ed);
	GG(a, b, c, d, x[13], S21, 0xa9e3e905);
	GG(d, a, b, c, x[2], S22, 0xfcefa3f8);
	GG(c, d, a, b, x[7], S23, 0x676f02d9);
	GG(b, c, d, a, x[12], S24, 0x8d2a4c8a);

	HH(a, b, c, d, x[5], S31, 0xfffa3942);
	HH(d, a, b, c, x[8], S32, 0x8771f681);
	HH(c, d, a, b, x[11], S33, 0x6d9d6122);
	HH(b, c, d, a, x[14], S34, 0xfde5380c);
	HH(a, b, c, d, x[1], S31, 0xa4beea44);
	HH(d, a, b, c, x[4], S32, 0x4bdecfa9);
	HH(c, d, a, b, x[7], S33, 0xf6bb4b60);
	HH(b, c, d, a, x[10], S34, 0xbebfbc70);
	HH(a, b, c, d, x[13], S31, 0x289b7ec6);
	HH(d, a, b, c, x[0], S32, 0xeaa127fa);
	HH(c, d, a, b, x[3], S33, 0xd4ef3085);
	HH(b, c, d, a, x[6], S34, 0x4881d05);
	HH(a, b, c, d, x[9], S31, 0xd9d4d039);
	HH(d, a, b, c, x[12], S32, 0xe6db99e5);
	HH(c, d, a, b, x[15], S33, 0x1fa27cf8);
	HH(b, c, d, a, x[2], S34, 0xc4ac5665);

	II(a, b, c, d, x[0], S41, 0xf4292244);
	II(d, a, b, c, x[7], S42, 0x432aff97);
	II(c, d, a, b, x[14], S43, 0xab9423a7);
	II(b, c, d, a, x[5], S44, 0xfc93a039);
	II(a, b, c, d, x[12], S41, 0x655b59c3);
	II(d, a, b, c, x[3], S42, 0x8f0ccc92);
	II(c, d, a, b, x[10], S43, 0xffeff47d);
	II(b, c, d, a, x[1], S44, 0x85845dd1);
	II(a, b, c, d, x[8], S41, 0x6fa87e4f);
	II(d, a, b, c, x[15], S42, 0xfe2ce6e0);
	II(c, d, a, b, x[6], S43, 0xa3014314);
	II(b, c, d, a, x[13], S44, 0x4e0811a1);
	II(a, b, c, d, x[4], S41, 0xf7537e82);
	II(d, a, b, c, x[11], S42, 0xbd3af235);
	II(c, d, a, b, x[2], S43, 0x2ad7d2bb);
	II(b, c, d, a, x[9], S44, 0xeb86d391);

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
}

__device__ __host__ void md5(const uchar* data, const uint size, uint result[4])
{
	uint state[4] = { 0x67452301, 0xefcdab89, 0x98badcfe, 0x10325476 }, i;

	for (i = 0; i + block_size <= size; i += block_size)
	{
		transform(state, data + i);
	}

	uint size_in_bits = size << 3;
	uchar buffer[block_size];

	memcpy(buffer, data + i, size - i);
	memcpy(buffer + size - i, padding, block_size - (size - i));
	memcpy(buffer + block_size - (2 * sizeof(uint)), &size_in_bits, sizeof(uint));

	transform(state, buffer);

	memcpy(result, state, 4 * sizeof(uint));
}
